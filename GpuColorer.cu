#include "hip/hip_runtime.h"
﻿
#include "GpuColorer.h"
#include "GraphAux.h"
#include <stdio.h>
#include <stdlib.h>


#include <hip/hip_runtime.h>
#include ""
#include <hip/hip_runtime.h>

extern "C" {
	#include "CpuColorer.h"
	#include "common.h"
}
#include "hip/device_functions.h"

#define THREADxBLOCK 128


Colorer* GpuColor(Graph* graph, int type) {
	Colorer* colorer;

	CHECK(hipMallocManaged(&colorer, sizeof(Colorer)));
	uint n = graph->nodeSize;
	colorer->uncoloredNodes = true;
	colorer->misNotFound = true;

	// hipMalloc for arrays of struct Coloring;
	CHECK(hipMallocManaged(&(colorer->coloring), n * sizeof(uint)));
	memset(colorer->coloring, 0, n * sizeof(uint));
	
	// allocate space on the GPU for the random states
	hiprandState_t* states;
	uint* weigths;
	uint* permutation;
	

	dim3 threads(THREADxBLOCK);
	dim3 blocks((graph->nodeSize + threads.x - 1) / threads.x, 1, 1);
	uint seed = 0;
	
	
	// start coloring (dyn. parall.)
	switch (type) {
	case 0: // LUBY 
		permutation = managedRandomPermutation(n);
		LubyColorer <<< 1, 1 >>> (colorer, graph, permutation);
		hipDeviceSynchronize();
		CHECK(hipFree(permutation));
		break;
	case 1: // JP
		//CHECK(hipMalloc((void**)&states, n * sizeof(hiprandState_t)));
		weigths = cpuInit(n);
		//init << < blocks, threads >> > (seed, states, weigths, n);
		//hipDeviceSynchronize();
		JPcolorer <<< 1, 1 >>> (colorer, graph, weigths);
		hipDeviceSynchronize();
		colorer->numOfColors = findMax(colorer, n);
		//hipFree(states);
		hipFree(weigths);
		break;
	case 2: // LDF
		//CHECK(hipMalloc((void**)&states, n * sizeof(hiprandState_t)));
		weigths = cpuInit(n);
		//init << < blocks, threads >> > (seed, states, weigths, n);
		//hipDeviceSynchronize();
		LDFcolorer <<< 1, 1 >>> (colorer, graph, weigths);
		hipDeviceSynchronize();
		colorer->numOfColors = findMax(colorer, n);
		//hipFree(states);
		hipFree(weigths);
		break;
	}
	
	return colorer;
}

/*
* Luby MIS colorer
*/
__global__ void LubyColorer(Colorer * colorer, Graph * graph, uint * permutation) {
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graph->nodeSize + threads.x - 1) / threads.x, 1, 1);

	colorer->numOfColors = 0;
	// loop on ISs covering the graph
	while (colorer->uncoloredNodes) {
		colorer->uncoloredNodes = false;
		colorer->numOfColors++;
		while (colorer->misNotFound) {
			colorer->misNotFound = false;
			LubyfindMIS <<< blocks, threads >>> (colorer, graph, permutation);
			hipDeviceSynchronize();
			RemoveNeighs <<< blocks, threads >>> (colorer, graph, permutation);
			hipDeviceSynchronize();
		}
		colorMIS <<< blocks, threads >>> (colorer, graph, permutation);
		hipDeviceSynchronize();
	}
}

__global__ void LubyfindMIS(Colorer* colorer, Graph* graph, uint* permutation) {
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	uint numColors = colorer->numOfColors;

	if (idx >= graph->nodeSize || colorer->coloring[idx] != 0) {
		return;
	}
		
	uint offset = graph->cumDegs[idx];
	uint deg = graph->cumDegs[idx + 1] - graph->cumDegs[idx];

	for (uint j = 0; j < deg; j++) {
		uint neighID = graph->neighs[offset + j];

		if ((colorer->coloring[neighID] == 0 || colorer->coloring[neighID] == -1) && (permutation[idx] < permutation[neighID])) {
			colorer->uncoloredNodes = true;
			colorer->misNotFound = true;
			return;
		} 
		
	}
	colorer->coloring[idx] = -1;
	return;
}

__global__ void RemoveNeighs(Colorer* colorer, Graph* graph, uint* weights) {
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (idx >= graph->nodeSize || colorer->coloring[idx] != 0) {
		return;
	}
	
	uint offset = graph->cumDegs[idx];
	uint deg = graph->cumDegs[idx + 1] - graph->cumDegs[idx];
		
	for (uint j = 0; j < deg; j++) {
		uint neighID = graph->neighs[offset + j];

		if (colorer->coloring[neighID] == -1) {
			colorer->coloring[idx] = -2;
			return;
		}
	}

}

__global__ void colorMIS(Colorer* colorer, Graph* graph, uint* weights) {

	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	colorer->misNotFound = true;
	if (colorer->coloring[idx] == -1 && idx < graph->nodeSize) {
		colorer->coloring[idx] = colorer->numOfColors;
	}
	else if (colorer->coloring[idx] == -2 && idx < graph->nodeSize){
		colorer->coloring[idx] = 0;
	}
	else {
		return;
	}

}

uint* managedRandomPermutation(uint n) {
	uint* permutation;
	CHECK(hipMallocManaged(&permutation, n * sizeof(uint)));
	//uint* permutation = (uint*)malloc(n * sizeof(uint));
	// initial range of numbers
	for (int i = 0;i < n;++i) {
		permutation[i] = i + 1;
	}
	// shuffle
	for (int i = n - 1; i >= 0; --i) {
		//generate a random number [0, n-1]
		int j = rand() % (i + 1);

		//swap the last element with element at random index
		int temp = permutation[i];
		permutation[i] = permutation[j];
		permutation[j] = temp;
	}
	return permutation;
}

/**
 *  this GPU kernel takes an array of states, and an array of ints, and puts a random int into each
 */
__global__ void init(uint seed, hiprandState_t* states, uint* numbers, uint n) {
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > n)
		return;
	hiprand_init(seed, idx, 0, &states[idx]);
	numbers[idx] = hiprand(&states[idx]);
}

uint* cpuInit(uint n) {
	uint* numbers;
	srand(time(NULL));
	CHECK(hipMallocManaged(&numbers, n * sizeof(uint)));
	for (int i = 0; i < n; i++) {
		numbers[i] = rand();
	}
	return numbers;
}

/**
 * Luby IS & Lones−Plassmann colorer
 */
__global__ void JPcolorer(Colorer* colorer, Graph* graph, uint* weights) {
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graph->nodeSize + threads.x - 1) / threads.x, 1, 1);

	// loop on ISs covering the graph
	colorer->numOfColors = 0;
	while (colorer->uncoloredNodes) {
		colorer->uncoloredNodes = false;
		colorer->numOfColors++;
		JPfindIS << < blocks, threads >> > (colorer, graph, weights);
		hipDeviceSynchronize();
		colorIsWithMin << < blocks, threads >> > (colorer, graph, weights);
		hipDeviceSynchronize();
	}
}

__global__ void JPfindIS(Colorer* colorer, Graph* graph, uint* weights) {
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;
	//uint numColors = colorer->numOfColors;
	if (idx >= graph->nodeSize)
		return;

	if (colorer->coloring[idx])
		return;

	uint offset = graph->cumDegs[idx];
	uint deg = graph->cumDegs[idx + 1] - graph->cumDegs[idx];

	for (uint j = 0; j < deg; j++) {
		uint neighID = graph->neighs[offset + j];
		uint degNeigh = weights[neighID];

		if (colorer->coloring[neighID] <= 0 && ((weights[idx] < weights[neighID]) || ((weights[idx] == weights[neighID]) && idx < neighID))) {
			colorer->uncoloredNodes = true;
			return;
		}
	}
	colorer->coloring[idx] = -1;
}


/**
*LDF colorer
*/
__global__ void LDFcolorer(Colorer* colorer, Graph* graph, uint* weights) {
	dim3 threads(THREADxBLOCK);
	dim3 blocks((graph->nodeSize + threads.x - 1) / threads.x, 1, 1);

	// loop on ISs covering the graph
	colorer->numOfColors = 0;
	while (colorer->uncoloredNodes) {
		colorer->uncoloredNodes = false;
		colorer->numOfColors++;
		LDFfindIS <<< blocks, threads >>> (colorer, graph, weights);
		hipDeviceSynchronize();
		colorIsWithMin <<< blocks, threads >>> (colorer, graph, weights);
		hipDeviceSynchronize();
	}
	
}

/**
 * find an IS
 */
__global__ void LDFfindIS(Colorer* colorer, Graph* graph, uint* weights) {
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= graph->nodeSize)
		return;

	if (colorer->coloring[idx])
		return;

	uint offset = graph->cumDegs[idx];
	uint deg = graph->cumDegs[idx + 1] - graph->cumDegs[idx];

	for (uint j = 0; j < deg; j++) {

		uint neighID = graph->neighs[offset + j];
		uint degNeigh = graph->cumDegs[neighID + 1] - graph->cumDegs[neighID];


		if (colorer->coloring[neighID] <= 0 && ((deg < degNeigh) || ((deg == degNeigh) && weights[idx] < weights[neighID]))) {
			colorer->uncoloredNodes = true;
			return;
		}
	}

	colorer->coloring[idx] = -1;

}

/**
 * color an IS
 */
__global__ void colorIsWithMin(Colorer* colorer, Graph* graph, uint* weights) {
	
	uint idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (colorer->coloring[idx] == -1 && idx < graph->nodeSize) {

		uint offset = graph->cumDegs[idx];
		uint deg = graph->cumDegs[idx + 1] - graph->cumDegs[idx];
		
		//find lowest color available
		int lowest = 0;

		for (uint k = 1; k <= deg + 1; k++) { // <= because there are at most n+1 colors, we start from 0 because tha 0 is for non-colored
			bool candidate = true;
			lowest = k;
			for (uint j = 0; j < deg; j++) {
				uint neighID = graph->neighs[offset + j];
				if (colorer->coloring[neighID] == k) {
					candidate = false;
					break;
				}
			}
			if (candidate) {
				break;
			}
		}

		colorer->coloring[idx] = lowest;

	}
	else {
		return;
	}

}

int findMax(Colorer* colorer, int n) {
	int max = 0;
	int index = 0;
	for (int i = 0; i < n; i++) {
		if (colorer->coloring[i] > max) {
			max = colorer->coloring[i];
			index = i;
		}
	}
	//printf("max %d at index %d\n", max, index);
	return max;
}



